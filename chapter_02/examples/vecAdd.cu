
#include <hip/hip_runtime.h>
#include <cmath>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>

__global__ void add(float *a, float *b, float *c, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

void vecAdd(float *ah, float *bh, float *ch, int n) {
  int size = n * sizeof(float);
  float *ad, *bd, *cd;

  hipMalloc((void **)&ad, size);
  hipMalloc((void **)&bd, size);
  hipMalloc((void **)&cd, size);

  hipMemcpy(ad, ah, size, hipMemcpyHostToDevice);
  hipMemcpy(bd, bh, size, hipMemcpyHostToDevice);

  add<<<ceil(n / 4.0), 4>>>(ad, bd, cd, size);

  hipMemcpy(ch, cd, size, hipMemcpyDeviceToHost);
  hipFree(ah);
  hipFree(ch);
  hipFree(bh);
}

int main() {
  printf("Adding vecs: {1, 2, 3} + {4, 3, 2}\n");
  float a[] = {1, 2, 3};
  float b[] = {4, 3, 2};
  float c[] = {0, 0, 0};

  vecAdd(a, b, c, 3);
  printf("Result: { ");
  for (int i = 0; i < 3; i++) {
    printf("%.1f", c[i]);
    if (i < 2)
      printf(", ");
  }
  printf(" }\n");

  return 0;
}